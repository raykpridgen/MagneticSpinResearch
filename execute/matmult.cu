#include <iostream>
#include <hip/hip_runtime.h>

// Determines size of one dimension of the matrix
#define N 4


// Kernel for matrix multiplication
__global__ void matMul(const float *A, const float *B, float *C, int n)
{
    // Define position within GPU for each thread to operate on
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // If array is in bounds
    if (row < n && col < n)
    {
        float sum = 0.0f;
        // For dimensions of square array
        for (int k = 0; k < n; k++)
        {
            // Sum across row A, col B          
            sum += A[row * n + k] * B[k * n + col];
        }
        // Place sum in a location on a new matrix
        C[row * n + col] = sum;
    }
}

int main()
{
    // Calculate size (n x n) matrix * size of float
    int size = N * N * sizeof(float);

    // Allocate host (CPU) memory, where the data starts at
    float h_A[N * N], h_B[N * N], h_C[N * N];

    // Initialize matrices A and B - fill the data within the host memory
    for (int i = 0; i < N * N; i++)
    {
        // Matrix A gets 0 - N*N for each entry
        h_A[i] = static_cast<float>(i);
        // Matrix B gets 0 - N repeating for each entry
        h_B[i] = static_cast<float>(i % N);
    }

    // Allocate device memory - Now set up the data bounds for the GPU itself
    float *d_A, *d_B, *d_C;
    // Notice pointers and references, d_X points to portion of GPU ?
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy host data to device - CPU -> GPU
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch kernel: organize threads in 2D grid
    // 16 x 16 is a design decision
    // Min 32 threads executed at once, so block sizes mult of 32. 16 x 16 = 256 = 8 warps
    // dim3 is a struct that organizes up to 3D blocks
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + 15) / 16, (N + 15) / 16);
    // <<<>>> is kernel launch syntax, () is kernel parameters
    // <<<>>> thread specs, () regular parameters
    matMul<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result back to host - GPU -> CPU
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Print result matrix C
    std::cout << "Result matrix C:\n";
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            std::cout << h_C[i * N + j] << " ";
        }
        std::cout << "\n";
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}